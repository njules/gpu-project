#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include "weights_vectors.h"
#include "input_vector.h"
#include "sigmoid.h"

#include <hip/hip_runtime.h>


#define imgsize 32
#define filter_size 5


__global__ void convolution(int a_width, int b_width, int channel_in, int channel_out,
                long double *matrix_a, //[channel_in][a_width][a_width]
                long double *matrix_b, //[channel_out][channel_in][b_width][b_width]
                long double *matrix_c, //[channel_out][a_width - b_width + 1][a_width - b_width + 1]
                long double *bias); //[channel_out]*/

__global__ void avgpool();

__global__ void fully_connected();

__global__ void sigmoid();

void softmax();

int main(){

//KERNEL CONFIGURATION

int threads = 5;
dim3 block(threads); //threads
dim3 grid(6); //blocks

hipSetDevice(0);

//LAYER 1

long double *dev_input_conv1, *dev_matrix_conv1, *dev_bias1, *dev_output_conv1;

long double *output_conv1;

output_conv1 = (long double*)malloc(5*5*3*6 * sizeof(long double));

// Host to Device

hipMalloc( (void**)&dev_input_conv1, 32*32*3 * sizeof(long double) );
hipMalloc( (void**)&dev_matrix_conv1, 5*5*3*6 * sizeof(long double) );
hipMalloc( (void**)&dev_output_conv1, 28*28*6 * sizeof(long double) );
hipMalloc( (void**)&dev_bias1, 6 * sizeof(long double) );

hipMemcpy( dev_input_conv1, input, 32*32*3 * sizeof(long double), hipMemcpyHostToDevice);
hipMemcpy( dev_matrix_conv1, conv1_weight, 5*5*3*6 * sizeof(long double), hipMemcpyHostToDevice);
hipMemcpy( dev_matrix_conv1, conv1_weight, 6 * sizeof(long double), hipMemcpyHostToDevice);

//kernel
convolution<<<grid,block>>>(32,5,3,6,dev_input_conv1,dev_matrix_conv1,dev_output_conv1,dev_bias1);

// Device to Host
hipMemcpy( output_conv1 , dev_output_conv1, 5*5*3*6 * sizeof(long double), hipMemcpyDeviceToHost);

// Freeing Space

hipFree(dev_input_conv1);
hipFree(dev_matrix_conv1);
hipFree(dev_output_conv1);
hipFree(dev_bias1);


//LAYER 2








hipDeviceReset();

}



long double sigmoidl(long double n) {

    return (1 / (1 + powf(EULER_NUMBER_L, -n)));

}

__global__ void convolution(int a_width, int b_width, int channel_in, int channel_out,
                long double *matrix_a, //[channel_in][a_width][a_width]
                long double *matrix_b, //[channel_out][channel_in][b_width][b_width]
                long double *matrix_c, //[channel_out][a_width - b_width + 1][a_width - b_width + 1]
                long double *bias){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	long double res = 0;
	
	__shared__ extern int s[];

	//to shared memory

	if(idx >= a_width)return;
	
	/*s[threadIdx.x+m/2] = a[idx];
	
	if(idx < m/2){
		s[threadIdx.x] = 0;
	}
	if(idx > n-m/2-1){
		s[threadIdx.x+m] = 0;
		
	}*/


	
	__syncthreads();

	//start computation
	
	/*if((threadIdx.x > blockDim.x - m/2 - 1 && idx < n - m/2) || (threadIdx.x < m/2 && idx > m/2)){ //its border use global
	
		for(int i = 0; i < m; i++){
			res += a[idx+i-m/2] * b[i];
		}
	
	}
	else{//its inner use shared
	
		for(int i = 0; i < m; i++){
			res += s[threadIdx.x+i] * b[i];
			
		}
	
	}*/



	//to output
	
	matrix_c[idx] = res;

}